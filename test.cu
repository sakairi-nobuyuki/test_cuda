
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


#define NN 200

__global__
void sum_array (double *array_1, double *array_2, double *array_3, int n_array) {
    int i, j, n;
    i = blockIdx.x * blockDim.x + threadIdx.x;

    n = 10;

    for (j = 0; j < n; j++) {
        if (i < NN)  array_3[i] = array_1[i] + array_3[i];        
    }
    
}

void initialize_array (double *array, int size) {
    int i;

//    for (i = 0; i < NN; i++)  array[i] = (double) rand ();
    for (i = 0; i < NN; i++)  array[i] = 1.0;

}

void print_result (double *array, int n) {
    int i;


    for (i = 0; i < n; i++)  printf ("%.0lf ", array[i]);
    printf ("\n");

}

int main () {
    double *array_1, *array_2, *array_3;
    double *d_array_1, *d_array_2, *d_array_3;
    size_t n_bytes = NN * sizeof (double);
    dim3 Grid, Block;

    Grid.x = NN / 196 + 1;
    Block.x = 196;

    printf ("hoge\n");

    array_1 = (double *) malloc (n_bytes);
    array_2 = (double *) malloc (n_bytes);
    array_3 = (double *) malloc (n_bytes);

    printf ("memory allocation finished\n");

    initialize_array (array_1, n_bytes);
    initialize_array (array_2, n_bytes);
    initialize_array (array_3, n_bytes);

    printf ("initialize memory\n");


    printf ("cuda memory allocation\n");

    hipMalloc ((void**)&d_array_1, n_bytes);
    hipMalloc ((void**)&d_array_2, n_bytes);
    hipMalloc ((void**)&d_array_3, n_bytes);

    printf ("cuda memory allocation finished\n");


    printf ("cuda memory copy\n");

    hipMemcpy (d_array_1, array_1, n_bytes, hipMemcpyHostToDevice);
    hipMemcpy (d_array_2, array_2, n_bytes, hipMemcpyHostToDevice);
    hipMemcpy (d_array_3, array_3, n_bytes, hipMemcpyHostToDevice);

    printf ("cuda memory copy finished\n");

    printf ("inp array1\n");
    print_result (array_1, NN);
    printf ("inp array2\n");
    print_result (array_2, NN);
    printf ("inp array3\n");
    print_result (array_3, NN);

    printf ("start kernel function\n");

    sum_array<<<Grid, Block>>> (d_array_1, d_array_2, d_array_3, n_bytes);

    hipDeviceSynchronize();
    printf ("end kernel function\n");

    hipMemcpy (array_3, d_array_3, n_bytes, hipMemcpyDeviceToHost);

    printf ("res array3\n");
    print_result (array_3, NN);

    return 0;
}